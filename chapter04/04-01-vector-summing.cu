#include "hip/hip_runtime.h"
#include "../common/book.h"
#define N 100000

__global__ void add(int *a, int *b, int *c) {
	int tid = blockIdx.x;
	if (tid < N) {
		c[tid] = a[tid] + b[tid];
	}
}
int main(void) {
	int a[N], b[N], c[N];
	int *dev_a, *dev_b, *dev_c;

	//��GPU�Ϸ����ڴ�
	HANDLE_ERROR(hipMalloc((void**)&dev_a, N * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dev_b, N * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dev_c, N * sizeof(int)));

	//��CPU��Ϊ����a��b��ֵ
	for (int i = 0; i < N; i++) {
		a[i] = -i;
		b[i] = i * i;
	}
	//������a��b���Ƶ�GPU
	HANDLE_ERROR(hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_c, c, N * sizeof(int), hipMemcpyHostToDevice));

	add <<<N, 1 >>> (dev_a, dev_b, dev_c);
	
	//������c��GPU���Ƶ�CPU
	HANDLE_ERROR(hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost));

	for (int i = 0; i < N; i++) {
		printf("%d + %d = %d\n", a[i], b[i], c[i]);
	}

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	return 0;
}
