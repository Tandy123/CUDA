#include "hip/hip_runtime.h"
#include "../common/book.h"
#include "../common/cpu_bitmap.h"

#define DIM 1000
//struct hipComplex {
//	float r;
//	float i;
//	hipComplex(float a, float b) :r(a), i(b) {}
//	float magnitude2(void) { return r*r + i*i;}
//	hipComplex operator*(const hipComplex& a) {
//		return hipComplex(r * a.r - i * a.i, i * a.r + r * a.i);
//	}
//	hipComplex operator+(const hipComplex& a) {
//		return hipComplex(r + a.r, i + a.i);
//	}
//};

struct hipComplex {
	float r;
	float i;
	__device__ hipComplex(float a, float b) :r(a), i(b) {}
	__device__ float magnitude2(void) {
		return r*r + i*i;
	}
	__device__ hipComplex operator*(const hipComplex& a) {
		return hipComplex(r * a.r - i * a.i, i * a.r + r * a.i);
	}
	__device__ hipComplex operator+(const hipComplex& a) {
		return hipComplex(r + a.r, i + a.i);
	}
};

//int julia(int x, int y) {
//	const float scale = 1.5;
//	float jx = scale * (float)(DIM / 2 - x) / (DIM / 2);
//	float jy = scale * (float)(DIM / 2 - y) / (DIM / 2);
//
//	hipComplex c(-0.8, 0.156);
//	hipComplex a(jx, jy);
//
//	int i = 0;
//	for (i = 0; i <= 200; i++) {
//		a = a * a + c;
//		if (a.magnitude2() > 1000)
//			return 0;
//	}
//	return 1;
//}

__device__ int julia(int x, int y) {
	const float scale = 1.5;
	float jx = scale * (float)(DIM / 2 - x) / (DIM / 2);
	float jy = scale * (float)(DIM / 2 - y) / (DIM / 2);

	hipComplex c(-0.8, 0.156);
	hipComplex a(jx, jy);

	int i = 0;
	for (i = 0; i <= 200; i++) {
		a = a * a + c;
		if (a.magnitude2() > 1000)
			return 0;
	}
	return 1;
}

//void kernel(unsigned char *ptr) {
//	for (int y = 0; y < DIM; y++) {
//		for (int x = 0; x < DIM; x++) {
//			int offset = x + y * DIM;
//
//			int juliaValue = julia(x, y);
//			ptr[offset * 4 + 0] = 255 * juliaValue;
//			ptr[offset * 4 + 1] = 0;
//			ptr[offset * 4 + 2] = 0;
//			ptr[offset * 4 + 3] = 255;
//		}
//	}
//}

__global__ void kernel(unsigned char *ptr) {
	int x = blockIdx.x;
	int y = blockIdx.y;
	int offset = x + y * gridDim.x;

	int juliaValue = julia(x, y);
	ptr[offset * 4 + 0] = 255 * juliaValue;
	ptr[offset * 4 + 1] = 0;
	ptr[offset * 4 + 2] = 0;
	ptr[offset * 4 + 3] = 255;
}

int main() {

	//	CPUBitmap bitmap(DIM, DIM);
	//	unsigned char* ptr = bitmap.get_ptr();
	//
	//	kernel(ptr);
	//
	//	bitmap.display_and_exit();

	CPUBitmap bitmap(DIM, DIM);
	unsigned char* dev_bitmap;
	HANDLE_ERROR(hipMalloc((void**)&dev_bitmap, bitmap.image_size()));

	dim3 grid(DIM, DIM);
	kernel << <grid, 1 >> > (dev_bitmap);

	HANDLE_ERROR(hipMemcpy(bitmap.get_ptr(), dev_bitmap, bitmap.image_size(), hipMemcpyDeviceToHost));

	bitmap.display_and_exit();
	HANDLE_ERROR(hipFree(dev_bitmap));
	
}