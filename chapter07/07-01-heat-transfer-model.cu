#include "hip/hip_runtime.h"
#include "../common/book.h"
#include "../common/cpu_anim.h"

#define DIM 1024
#define PI 3.1415926535897932f
#define MAX_TEMP 1.0f
#define MIN_TEMP 0.0001f
#define SPEED 0.25f

__global__ void blend_kernel(float *outSrc, const float *inSrc) {
	// map from threadIdx/blockIdx to pixel position
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;

	int left = offset - 1;
	int right = offset + 1;
	if (x == 0) left++;
	if (x == DIM - 1) right--;

	int top = offset - DIM;
	int bottom = offset + DIM;
	if (y == 0) top += DIM;
	if (y == DIM - 1) bottom -= DIM;

	outSrc[offset] = inSrc[offset] + SPEED * (inSrc[top] + inSrc[bottom] + inSrc[left] + inSrc[right] - inSrc[offset] * 4);
}

__global__ void copy_const_kernel(float *iptr, const float *cptr) {
	// map from threadIdx/blockIdx to pixel position
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;

	if (cptr[offset] != 0) iptr[offset] = cptr[offset];
}

// globals needed by the update routine
struct DataBlock {
	unsigned char	*output_bitmap;
	float			*dev_inSrc;
	float			*dev_outSrc;
	float			*dev_constSrc;
	CPUAnimBitmap	*bitmap;
	hipEvent_t		start, stop;
	float			totalTime;
	float			frames;
};

void anim_gpu(DataBlock *d, int ticks) {
	HANDLE_ERROR(hipEventRecord(d->start, 0));
	dim3	blocks(DIM / 16, DIM / 16);
	dim3	threads(16, 16);
	CPUAnimBitmap	*bitmap = d->bitmap;

	for (int i = 0; i<300; i++) {
		copy_const_kernel << <blocks, threads >> >(d->dev_inSrc, d->dev_constSrc);
		blend_kernel << <blocks, threads >> >(d->dev_outSrc, d->dev_inSrc);
		swap(d->dev_inSrc, d->dev_outSrc);
	}
	float_to_color << <blocks, threads >> >(d->output_bitmap, d->dev_inSrc);

	HANDLE_ERROR(hipMemcpy(bitmap->get_ptr(), d->output_bitmap, bitmap->image_size(), hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipEventRecord(d->stop, 0));
	HANDLE_ERROR(hipEventSynchronize(d->stop));
	float elapsedTime;
	HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, d->start, d->stop));

	d->totalTime += elapsedTime;
	++d->frames;
	printf("Average time per frame: %3.1f ms\n", d->totalTime / d->frames);
}

void anim_exit(DataBlock *d) {
	hipFree(d->dev_inSrc);
	hipFree(d->dev_outSrc);
	hipFree(d->dev_constSrc);

	HANDLE_ERROR(hipEventDestroy(d->start));
	HANDLE_ERROR(hipEventDestroy(d->stop));
}

int main(void) {
	DataBlock	data;
	CPUAnimBitmap bitmap(DIM, DIM, &data);
	data.bitmap = &bitmap;
	data.totalTime = 0;
	data.frames = 0;
	HANDLE_ERROR(hipEventCreate(&data.start));
	HANDLE_ERROR(hipEventCreate(&data.stop));

	HANDLE_ERROR(hipMalloc((void**)&data.output_bitmap, bitmap.image_size()));

	// assume float == 4 chars in size (ie., rgba)
	HANDLE_ERROR(hipMalloc((void**)&data.dev_inSrc, bitmap.image_size()));
	HANDLE_ERROR(hipMalloc((void**)&data.dev_outSrc, bitmap.image_size()));
	HANDLE_ERROR(hipMalloc((void**)&data.dev_constSrc, bitmap.image_size()));

	float *temp = (float*)malloc(bitmap.image_size());
	for (int i = 0; i<DIM*DIM; i++) {
		temp[i] = 0;
		int x = i % DIM;
		int y = i / DIM;
		if ((x>300) && (x<600) && (y>310) && (y<601))
			temp[i] = MAX_TEMP;
	}
	temp[DIM * 100 + 100] = (MAX_TEMP + MIN_TEMP) / 2;
	temp[DIM * 700 + 100] = MIN_TEMP;
	temp[DIM * 300 + 300] = MIN_TEMP;
	temp[DIM * 200 + 700] = MIN_TEMP;
	for (int y = 800; y<900; y++) {
		for (int x = 400; x<500; x++) {
			temp[x + y*DIM] = MIN_TEMP;
		}
	}
	HANDLE_ERROR(hipMemcpy(data.dev_constSrc, temp, bitmap.image_size(), hipMemcpyHostToDevice));

	for (int y = 800; y<DIM; y++) {
		for (int x = 0; x<200; x++) {
			temp[x + y*DIM] = MAX_TEMP;
		}
	}
	HANDLE_ERROR(hipMemcpy(data.dev_inSrc, temp, bitmap.image_size(), hipMemcpyHostToDevice));

	free(temp);

	bitmap.anim_and_exit((void(*)(void*, int))anim_gpu, (void(*)(void*))anim_exit);
}



