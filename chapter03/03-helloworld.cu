#include "hip/hip_runtime.h"
#include<iostream>
#include "../common/book.h"

__global__ void add(int a, int b, int *c) {
	*c = a + b;
}

int main()
{
	int c;
	int *dev_c;
	HANDLE_ERROR(hipMalloc((void**)&dev_c, sizeof(int)));
	add <<<1, 1 >>> (2, 7, dev_c);
	//printf("2 + 7 = %d\n", *dev_c);//error
	HANDLE_ERROR(hipMemcpy(&c,
		dev_c, 
		sizeof(int),
		hipMemcpyDeviceToHost));
	printf("2 + 7 = %d\n", c);

	int count;
	HANDLE_ERROR(hipGetDeviceCount(&count));
	printf("Device Count: %d\n", count);
	hipDeviceProp_t prop;
	for (int i = 0; i < count; i++) {
		HANDLE_ERROR(hipGetDeviceProperties(&prop, i));
		printf("Name: %s\n", prop.name);
	}
	int dev;
	HANDLE_ERROR(hipGetDevice(&dev));
	printf("ID of current CUDA device: %d\n", dev);

	memset(&prop, 0, sizeof(hipDeviceProp_t));
	prop.major = 1;
	prop.minor = 3;
	HANDLE_ERROR(hipChooseDevice(&dev, &prop));
	printf("ID of CUDA device closest to revision 1.3: %d\n", dev);
	HANDLE_ERROR(hipSetDevice(dev));
	hipFree(dev_c);
	return 0;
}