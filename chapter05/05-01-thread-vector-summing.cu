#include "hip/hip_runtime.h"
/*********************************************************************************
*FileName:  05-01-thread-vector-summing.cu
*Author:  Tandy
*Date:  2017-07-04
*Description:  ʵ�ֲ��޳��ȵ�ʸ�����
**********************************************************************************/

#include "../common/book.h"
#define N (33 * 1024)

__global__ void add(int *a, int *b, int *c) {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	while (tid < N) {
		c[tid] = a[tid] + b[tid];
		tid = tid + blockDim.x*gridDim.x;
	}
}
int main(void) {
	int a[N], b[N], c[N];
	int *dev_a, *dev_b, *dev_c;

	//��GPU�Ϸ����ڴ�
	HANDLE_ERROR(hipMalloc((void**)&dev_a, N * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dev_b, N * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dev_c, N * sizeof(int)));

	//��CPU��Ϊ����a��b��ֵ
	for (int i = 0; i < N; i++) {
		a[i] = i;
		b[i] = i * i;
	}
	//������a��b���Ƶ�GPU
	HANDLE_ERROR(hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_c, c, N * sizeof(int), hipMemcpyHostToDevice));

	add <<<128, 128>>> (dev_a, dev_b, dev_c);

	//������c��GPU���Ƶ�CPU
	HANDLE_ERROR(hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost));
	
	//��֤GPUȷʵ���������Ҫ��Ĺ���
	bool success = true;
	for (int i = 0; i < N; i++) {
		if ((a[i] + b[i]) != c[i]) {
			printf("Error: %d + %d != %d\n", a[i], b[i], c[i]);
			success = false;
		}
	}
	if (success) {
		printf("We did it!\n");
	 }

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	return 0;
}
