#include "hip/hip_runtime.h"
#include "../common/book.h"

#define imin(a, b) (a < b?a:b)

const int N = 100 * 1024;
const int threadsPerBlock = 256;
const int blocksPerGrid = imin(32, (N + threadsPerBlock - 1)/threadsPerBlock);

__global__ void dot(float *a, float *b, float *c) {
	__shared__ float cache[threadsPerBlock];
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int cacheIndex = threadIdx.x;

	float temp = 0;
	while (tid < N) {
		temp += a[tid] * b[tid];
		tid += blockDim.x * gridDim.x;
	}

	cache[cacheIndex] = temp;
	//���߳̿��е��߳̽���ͬ��
	__syncthreads();
	//���ڹ�Լ������˵�����´���Ҫ��threadsPerBlock������2��ָ��
	int i = blockDim.x / 2;
	while (i != 0) {
		if (cacheIndex < i) {
			cache[cacheIndex] += cache[cacheIndex + i];
		}
		__syncthreads();
		i /= 2;
	}
	if (cacheIndex == 0) {
		c[blockIdx.x] = cache[0];
	}
}

int main() {
	float *a, *b, c, *partial_c;
	float *dev_a, *dev_b, *dev_partial_c;

	a = (float*)malloc(N * sizeof(float));
	b = (float*)malloc(N * sizeof(float));
	partial_c = (float*)malloc(blocksPerGrid * sizeof(float));

	HANDLE_ERROR(hipMalloc((void**)&dev_a, N * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&dev_b, N * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&dev_partial_c, N * sizeof(float)));

	for (int i = 0; i < N; ++i) {
		a[i] = i;
		b[i] = i * 2;
	}

	HANDLE_ERROR(hipMemcpy(dev_a, a, N * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_b, b, N * sizeof(float), hipMemcpyHostToDevice));

	dot <<<blocksPerGrid, threadsPerBlock >>> (dev_a, dev_b, dev_partial_c);

	HANDLE_ERROR(hipMemcpy(partial_c, dev_partial_c, blocksPerGrid * sizeof(float), hipMemcpyDeviceToHost));//ע�����￪�ٿռ�Ĵ�С

	c = 0;
	for (int i = 0; i < blocksPerGrid; i++) {
		c += partial_c[i];
	}

#define sum_squares(x) (x*(x + 1)* (2 * x + 1)/6)
	printf("Does GPU value %.6g = %.6g?\n", c, 2 * sum_squares((float)(N - 1)));

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_partial_c);

	free(a);
	free(b);
	free(partial_c);
}