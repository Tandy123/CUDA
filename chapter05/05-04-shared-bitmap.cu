#include "hip/hip_runtime.h"
#include "../common/book.h"
#include "../common/cpu_bitmap.h"

#define DIM 1024
#define PI 3.1415926535897932f

__global__ void kernel(unsigned char* ptr) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;

	__shared__ float shared[16][16];

	const float period = 128.0f;

	shared[threadIdx.x][threadIdx.y] =
		255 * (sinf(x * 2.0f * PI / period) + 1.0f) *
		(sinf(y * 2.0f * PI / period) + 1.0f) / 4.0f;

	//__syncthreads();

	ptr[offset * 4 + 0] = 0;
	ptr[offset * 4 + 1] = shared[15 - threadIdx.x][15 - threadIdx.y];
	//ptr[offset * 4 + 1] = shared[threadIdx.x][threadIdx.y];//���߳��໥����
	ptr[offset * 4 + 2] = 0;
	ptr[offset * 4 + 3] = 255;
}

int main(void) {
	CPUBitmap bitmap(DIM, DIM);
	unsigned char* dev_bitmap;

	HANDLE_ERROR(hipMalloc((void**)&dev_bitmap, bitmap.image_size()));

	dim3 grids(DIM / 16, DIM / 16);
	dim3 threads(16, 16);
	kernel << <grids, threads >> > (dev_bitmap);
	HANDLE_ERROR(hipMemcpy(bitmap.get_ptr(), dev_bitmap, bitmap.image_size(), hipMemcpyDeviceToHost));

	bitmap.display_and_exit();
	hipFree(dev_bitmap);
}