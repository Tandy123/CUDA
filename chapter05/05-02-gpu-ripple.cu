#include "hip/hip_runtime.h"
#include "../common/book.h"
#include "../common/cpu_anim.h"

#define DIM 1024

struct DataBlock {
	unsigned char *dev_bitmap;
	CPUAnimBitmap *bitmap;
};

void cleanup(DataBlock *d) {
	hipFree(d->dev_bitmap);
}

__global__ void kernel(unsigned char* ptr, int ticks) {
	//��threadIdx��BlockIdxӳ�䵽����λ��
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;
	//int offset = y + x * blockDim.y * gridDim.y;//������offset��Ч

	float fx = x - DIM / 2;
	float fy = y - DIM / 2;
	float d = sqrtf(fx * fx + fy * fy);
	unsigned char grey = (unsigned char)(128.0f + 127.0f * cos(d / 10.0f - ticks / 7.0f) / (d / 10.0f + 1.0f));
	ptr[offset * 4 + 0] = grey;
	ptr[offset * 4 + 1] = grey;
	ptr[offset * 4 + 2] = grey;
	ptr[offset * 4 + 3] = 255;
}

void generate_frame(DataBlock *d, int ticks) {
	dim3 blocks(DIM/16, DIM/16);
	dim3 threads(16, 16);
	kernel <<<blocks, threads >>> (d->dev_bitmap, ticks);
	HANDLE_ERROR(hipMemcpy(d->bitmap->get_ptr(),
		d->dev_bitmap,
		d->bitmap->image_size(), 
		hipMemcpyDeviceToHost));
}

int main() {
	DataBlock data;
	CPUAnimBitmap bitmap(DIM, DIM, &data);
	data.bitmap = &bitmap;
	HANDLE_ERROR(hipMalloc((void**)&data.dev_bitmap, bitmap.image_size()));

	bitmap.anim_and_exit((void(*)(void*, int))generate_frame, (void(*)(void*))cleanup);
}